#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "VSimple.h"

#define size1 127

__global__ void virtualcounter(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;
	if (i < n)
		c[i] = a[i] + b[i];
}


__global__ void virtualcounter1(int *max, int *res)
{

	VSimple vsm;
	//vsm.SimpleCouter(max, res);
	vsm.Justtest(max, res);


}


int main()
{
	{
		int a[size1], b[size1], c[size1];
		int *d_a, *d_b, *d_c;



		hipMalloc(&d_a, size1 * sizeof(int));
		hipMalloc(&d_b, size1 * sizeof(int));
		hipMalloc(&d_c, size1 * sizeof(int));

		for (char i = 0; i < 127; i++)
		{
			a[i] = i;
			b[i] = i;
			c[i] = i;
		}

		hipMemcpy(d_a, a, size1 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, size1 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_c, c, size1 * sizeof(int), hipMemcpyHostToDevice);

		virtualcounter <<<1, size1 >>> (d_a, d_b, d_c, size1);

		hipMemcpy(c, d_c, size1 * sizeof(int), hipMemcpyDeviceToHost);
		for (char i = 0; i < 10; i++)
		{
			printf("\n  %i", c[i]);
		}


		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

	}



	{
		int *max, *res;
		int *d_max, *d_res;

		max = (int *)malloc(sizeof(int));
		res = (int *)malloc(20 * sizeof(int));


		hipMalloc(&d_max, sizeof(int));
		hipMalloc(&d_res, 20 * sizeof(int));


		*max = 10;
		for (char i = 0; i < 20; i++)
		{
			if (i < 10) res[i] = 0;
			else res[i] = -1;
		}




		hipMemcpy(d_max, max, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_res, res, 20 * sizeof(int), hipMemcpyHostToDevice);

		virtualcounter1 << <1, size1 >> > (d_max, d_res);
		for (char i = 0; i < 20; i++)			printf("%i   ", res[i]);


		hipMemcpy(res, d_res, 20 * sizeof(int), hipMemcpyDeviceToHost);
		printf("\n  ");
		for (char i = 0; i < 20; i++)			printf("%i   ", res[i]);


		free(max);
		free(res);

		hipFree(d_max);
		hipFree(d_res);

	}




}


